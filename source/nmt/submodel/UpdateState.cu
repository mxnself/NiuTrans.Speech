#include "hip/hip_runtime.h"
/* NiuTrans.NMT - an open-source neural machine translation system.
 * Copyright (C) 2024 NiuTrans Research. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "UpdateState.cuh"

namespace nmt {

#ifdef USE_CUDA

  __global__ void updateStateKernel(const float* const src,
                                    const int* const index,
                                    const UpdateStateParams params,
                                    const int split_size,
                                    float * const tgt) {
    // TODO: niutensor doesn't guarantee 128-bit aligned.
    // Use 128-bit loading after 128-bit aligned support.
    constexpr int elements_per_thread = 1;
    const int threads_per_block = blockDim.x;
    const int elements_per_block = elements_per_thread * threads_per_block;
    // split_size for current tb
    const int current_split_size = blockIdx.x == gridDim.x-1 ?
                                   params.seqlen * params.head_dim - blockIdx.x * split_size :
                                   split_size;
  
    const int src_hid = blockIdx.z;
    const int src_bid = index[blockIdx.y];
    const int src_split_id = blockIdx.x;
    const int src_hb_split_id = (src_hid * params.src_batch_size + src_bid) * params.seqlen * params.head_dim + src_split_id * split_size;
  
    const int tgt_hid = blockIdx.z;
    const int tgt_bid = blockIdx.y;
    const int tgt_split_id = blockIdx.x;
    const int tgt_hb_split_id = (tgt_hid * params.tgt_batch_size + tgt_bid) * params.seqlen * params.head_dim + tgt_split_id * split_size;
    #pragma unroll
    for(int i=threadIdx.x*elements_per_thread;i<current_split_size;i+=elements_per_block) {
      tgt[tgt_hb_split_id+i] = src[src_hb_split_id+i];
    }
  }

  void updateState(const XTensor* const src,
                   const XTensor* const index,
                   const struct UpdateStateParams params,
                   XTensor* const tgt) {
    CheckNTErrors(src != nullptr &&
                  index != nullptr &&
                  tgt != nullptr,
                  "Invalid tensor!");
    CheckNTErrors(src->dataType == X_FLOAT, "only support state with type X_FLOAT now!");
    CheckNTErrors(tgt->dataType == X_FLOAT, "only support state with type X_FLOAT now!");
    CheckNTErrors(index->dataType == X_INT, "index must be type X_INT!");
    CheckNTErrors(src->devID >= 0, "the state must be kept on the gpu!");
    CheckNTErrors(src->devID == tgt->devID, "the state must be kept on the same device!");
    CheckNTErrors((src->unitSize == tgt->unitSize), "Unmatched tensors!");

    int devID = src->devID;
    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    // shorthand for seqlen*head_dim
    const auto sh_num = params.seqlen * params.head_dim;

    // split along (L, H)
    // TODO: add heuristic (maybe)
    const int loads_per_block = UPDATE_STATE_LOADS_PER_BLOCK;
    const int max_split_size = GDevs.GPUs[devID].GPUMaxThreadNumPerBlock * loads_per_block;
    const int split_size = sh_num > max_split_size ? max_split_size : sh_num;
    const int num_split = (sh_num + split_size - 1) / split_size;

    dim3 blocks(num_split, params.tgt_batch_size, params.num_head);

    dim3 threads((split_size + loads_per_block - 1) / loads_per_block);

    updateStateKernel<<<blocks, threads>>>(static_cast<float*>(src->data),
                                           static_cast<int*>(index->data),
                                           params,
                                           split_size,
                                           static_cast<float*>(tgt->data));

    BacktoCudaDev(devID, devIDBackup);
  }
#endif // USE_CUDA

} /* end of the nmt namespace */